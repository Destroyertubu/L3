#include "hip/hip_runtime.h"
#include "bitpack_utils.cuh"
#include "L3_format.hpp"
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cmath>

// ModelType enum is defined in L3_format.hpp

// Partition metadata optimized for GPU (coalesced access)
struct PartitionMeta {
    int32_t start_idx;
    int32_t partition_len;
    int32_t model_type;
    int32_t delta_bits;
    double theta0;
    double theta1;
    int64_t bit_offset_base;
};

// Compressed data structure (compatible with original)
template<typename T>
struct CompressedDataOpt {
    // Metadata arrays (SoA layout)
    int32_t* d_start_indices;
    int32_t* d_end_indices;
    int32_t* d_model_types;
    double* d_model_params;  // [theta0, theta1, theta2, theta3] for each partition
    int32_t* d_delta_bits;
    int64_t* d_delta_array_bit_offsets;

    // Bit-packed delta array
    uint32_t* delta_array;

    // Optional: pre-unpacked deltas for maximum throughput
    int64_t* d_plain_deltas;

    // Metadata
    int num_partitions;
    int total_elements;
};

// Apply delta to predicted value
template<typename T>
__device__ __forceinline__ T applyDelta(T predicted, int64_t delta) {
    if constexpr (std::is_signed<T>::value) {
        return predicted + static_cast<T>(delta);
    } else {
        return static_cast<T>(static_cast<int64_t>(predicted) + delta);
    }
}

// ============================================================================
// OPTIMIZED KERNEL: Warp-cooperative decompression with shared memory staging
// ============================================================================
// Key optimizations:
// 1. Warp loads delta chunks cooperatively into shared memory
// 2. Reduced divergence via early model type specialization
// 3. Vectorized/coalesced memory access
// 4. Register tiling: process multiple elements per thread
// 5. Prefetching of next partition's data
// ============================================================================

// Shared memory buffer size: 256 words + 2 extra for cross-word boundary reads
// This ensures extractBitsFromShared can safely read word_idx+1 even at buffer end
constexpr int SMEM_BUFFER_WORDS = 258;  // 256 + 2 = 1032 bytes
constexpr int SMEM_USABLE_WORDS = 256;  // Actual words for data
constexpr int ELEMENTS_PER_WARP_LOAD = (SMEM_USABLE_WORDS * 32) / 8;  // Approx elements for 8-bit deltas

template<typename T>
__global__ void __launch_bounds__(256, 4)  // Tune for occupancy
decompressPartitionsOptimized(
    const CompressedDataOpt<T> compressed_data,
    T* __restrict__ output)
{
    // Shared memory for metadata and delta staging
    __shared__ PartitionMeta s_meta;
    __shared__ uint32_t s_delta_buffer[SMEM_BUFFER_WORDS];

    const int partition_idx = blockIdx.x;

    if (partition_idx >= compressed_data.num_partitions) {
        return;
    }

    // Load partition metadata (thread 0 only)
    if (threadIdx.x == 0) {
        s_meta.start_idx = compressed_data.d_start_indices[partition_idx];
        s_meta.partition_len = compressed_data.d_end_indices[partition_idx] -
                               compressed_data.d_start_indices[partition_idx];
        s_meta.model_type = compressed_data.d_model_types[partition_idx];
        s_meta.delta_bits = compressed_data.d_delta_bits[partition_idx];
        s_meta.bit_offset_base = compressed_data.d_delta_array_bit_offsets[partition_idx];

        const int params_idx = partition_idx * 4;
        s_meta.theta0 = compressed_data.d_model_params[params_idx];
        s_meta.theta1 = compressed_data.d_model_params[params_idx + 1];
    }

    __syncthreads();

    // Fast path: pre-unpacked deltas (highest throughput)
    if (compressed_data.d_plain_deltas != nullptr) {
        for (int local_idx = threadIdx.x; local_idx < s_meta.partition_len; local_idx += blockDim.x) {
            const int global_idx = s_meta.start_idx + local_idx;

            if (global_idx >= compressed_data.total_elements) continue;

            T final_value;

            if (s_meta.model_type == MODEL_DIRECT_COPY) {
                final_value = static_cast<T>(compressed_data.d_plain_deltas[global_idx]);
            } else {
                // Linear model prediction
                const double predicted = fma(s_meta.theta1, static_cast<double>(local_idx), s_meta.theta0);
                const T predicted_T = static_cast<T>(round(predicted));
                const int64_t delta = compressed_data.d_plain_deltas[global_idx];
                final_value = applyDelta(predicted_T, delta);
            }

            output[global_idx] = final_value;
        }
        return;
    }

    // Standard path: bit-packed deltas with warp-cooperative unpacking
    if (s_meta.delta_bits <= 0 || compressed_data.delta_array == nullptr) {
        // No deltas: just write predicted values
        for (int local_idx = threadIdx.x; local_idx < s_meta.partition_len; local_idx += blockDim.x) {
            const int global_idx = s_meta.start_idx + local_idx;
            if (global_idx >= compressed_data.total_elements) continue;

            T final_value;
            if (s_meta.model_type == MODEL_DIRECT_COPY) {
                final_value = static_cast<T>(0);
            } else {
                const double predicted = fma(s_meta.theta1, static_cast<double>(local_idx), s_meta.theta0);
                final_value = static_cast<T>(round(predicted));
            }

            output[global_idx] = final_value;
        }
        return;
    }

    // Warp-cooperative processing of bit-packed deltas
    const int warp_id = threadIdx.x / WARP_SIZE;
    const int lane_id = threadIdx.x % WARP_SIZE;
    const int num_warps = blockDim.x / WARP_SIZE;

    // Process partition in chunks that fit in shared memory
    const int max_elements_per_chunk = (SMEM_USABLE_WORDS * 32) / max(s_meta.delta_bits, 1);

    for (int chunk_start = warp_id * max_elements_per_chunk;
         chunk_start < s_meta.partition_len;
         chunk_start += num_warps * max_elements_per_chunk)
    {
        const int chunk_size = min(max_elements_per_chunk, s_meta.partition_len - chunk_start);

        if (chunk_size <= 0) break;

        // Calculate how many words to load
        const int words_needed = computeWordsNeeded(chunk_size, s_meta.delta_bits);
        const int words_to_load = min(words_needed + 1, SMEM_BUFFER_WORDS);  // +1 for cross-word reads

        // Warp-collective load into shared memory
        const int64_t chunk_bit_offset = s_meta.bit_offset_base +
                                        (static_cast<int64_t>(chunk_start) * s_meta.delta_bits);
        const int64_t chunk_word_offset = chunk_bit_offset >> 5;
        const int local_bit_offset_base = chunk_bit_offset & 31;

        warpLoadToShared<SMEM_BUFFER_WORDS>(
            compressed_data.delta_array,
            chunk_word_offset,
            s_delta_buffer,
            lane_id
        );

        __syncthreads();  // Ensure shared memory is populated

        // Each thread unpacks its elements from shared memory
        for (int i = lane_id; i < chunk_size; i += WARP_SIZE) {
            const int local_idx = chunk_start + i;
            const int global_idx = s_meta.start_idx + local_idx;

            if (global_idx >= compressed_data.total_elements) continue;

            // Extract delta from shared memory
            const int local_bit_offset = local_bit_offset_base + i * s_meta.delta_bits;
            const uint32_t extracted = extractBitsFromShared(s_delta_buffer, local_bit_offset, s_meta.delta_bits);

            T final_value;

            if (s_meta.model_type == MODEL_DIRECT_COPY) {
                // Direct copy: extracted value is the final value
                // For signed types, we need sign extension
                if constexpr (std::is_signed<T>::value) {
                    const int32_t signed_val = signExtend(extracted, s_meta.delta_bits);
                    final_value = static_cast<T>(signed_val);
                } else {
                    final_value = static_cast<T>(extracted);
                }
            } else {
                // Model-based: apply delta to prediction
                const int32_t delta = signExtend(extracted, s_meta.delta_bits);
                const double predicted = fma(s_meta.theta1, static_cast<double>(local_idx), s_meta.theta0);
                const T predicted_T = static_cast<T>(round(predicted));
                final_value = applyDelta(predicted_T, static_cast<int64_t>(delta));
            }

            // Coalesced write to global memory
            output[global_idx] = final_value;
        }

        __syncthreads();  // Prepare for next chunk
    }
}

// ============================================================================
// ALTERNATIVE KERNEL: Branchless template specialization for model types
// ============================================================================
// This version reduces divergence by creating separate code paths per model type

template<typename T, int MODEL_TYPE>
__global__ void __launch_bounds__(256, 4)
decompressPartitionsSpecialized(
    const CompressedDataOpt<T> compressed_data,
    T* __restrict__ output)
{
    __shared__ PartitionMeta s_meta;
    __shared__ uint32_t s_delta_buffer[SMEM_BUFFER_WORDS];

    const int partition_idx = blockIdx.x;

    if (partition_idx >= compressed_data.num_partitions) {
        return;
    }

    // Load metadata
    if (threadIdx.x == 0) {
        s_meta.start_idx = compressed_data.d_start_indices[partition_idx];
        s_meta.partition_len = compressed_data.d_end_indices[partition_idx] -
                               compressed_data.d_start_indices[partition_idx];
        s_meta.delta_bits = compressed_data.d_delta_bits[partition_idx];
        s_meta.bit_offset_base = compressed_data.d_delta_array_bit_offsets[partition_idx];

        const int params_idx = partition_idx * 4;
        s_meta.theta0 = compressed_data.d_model_params[params_idx];
        s_meta.theta1 = compressed_data.d_model_params[params_idx + 1];
    }

    __syncthreads();

    // Grid-stride loop with specialization
    for (int local_idx = threadIdx.x; local_idx < s_meta.partition_len; local_idx += blockDim.x) {
        const int global_idx = s_meta.start_idx + local_idx;

        if (global_idx >= compressed_data.total_elements) continue;

        int32_t delta = 0;

        if (s_meta.delta_bits > 0 && compressed_data.delta_array != nullptr) {
            const int64_t bit_offset = s_meta.bit_offset_base +
                                      (static_cast<int64_t>(local_idx) * s_meta.delta_bits);
            delta = extractDeltaDirect(compressed_data.delta_array, bit_offset, s_meta.delta_bits);
        }

        T final_value;

        if constexpr (MODEL_TYPE == MODEL_DIRECT_COPY) {
            final_value = static_cast<T>(delta);
        } else if constexpr (MODEL_TYPE == MODEL_LINEAR) {
            const double predicted = fma(s_meta.theta1, static_cast<double>(local_idx), s_meta.theta0);
            const T predicted_T = static_cast<T>(round(predicted));
            final_value = applyDelta(predicted_T, static_cast<int64_t>(delta));
        } else {
            // Fallback for other model types
            const double predicted = fma(s_meta.theta1, static_cast<double>(local_idx), s_meta.theta0);
            const T predicted_T = static_cast<T>(round(predicted));
            final_value = applyDelta(predicted_T, static_cast<int64_t>(delta));
        }

        output[global_idx] = final_value;
    }
}

// ============================================================================
// HOST API FUNCTIONS
// ============================================================================

template<typename T>
hipError_t launchDecompressOptimized(
    const CompressedDataOpt<T>& compressed_data,
    T* output,
    hipStream_t stream = 0)
{
    if (compressed_data.num_partitions == 0) {
        return hipSuccess;
    }

    const int num_blocks = compressed_data.num_partitions;
    const int threads_per_block = 256;

    decompressPartitionsOptimized<T><<<num_blocks, threads_per_block, 0, stream>>>(
        compressed_data,
        output
    );

    return hipGetLastError();
}

// Explicit instantiations for common types
template hipError_t launchDecompressOptimized<int32_t>(
    const CompressedDataOpt<int32_t>&, int32_t*, hipStream_t);
template hipError_t launchDecompressOptimized<int64_t>(
    const CompressedDataOpt<int64_t>&, int64_t*, hipStream_t);
template hipError_t launchDecompressOptimized<uint32_t>(
    const CompressedDataOpt<uint32_t>&, uint32_t*, hipStream_t);
template hipError_t launchDecompressOptimized<uint64_t>(
    const CompressedDataOpt<uint64_t>&, uint64_t*, hipStream_t);

// ============================================================================
// Wrapper for CompressedDataGLECO format (L3_codec.hpp compatibility)
// ============================================================================

/**
 * Wrapper to decompress using CompressedDataGLECO format
 *
 * This function converts CompressedDataGLECO to CompressedDataOpt and calls
 * the optimized decompression kernel.
 */
template<typename T>
void launchDecompressOptimized(
    const CompressedDataGLECO<T>* compressed,
    T* d_output,
    hipStream_t stream)
{
    // Convert CompressedDataGLECO to CompressedDataOpt
    CompressedDataOpt<T> opt_format;
    opt_format.d_start_indices = compressed->d_start_indices;
    opt_format.d_end_indices = compressed->d_end_indices;
    opt_format.d_model_types = compressed->d_model_types;
    opt_format.d_model_params = compressed->d_model_params;
    opt_format.d_delta_bits = compressed->d_delta_bits;
    opt_format.d_delta_array_bit_offsets = compressed->d_delta_array_bit_offsets;
    opt_format.delta_array = compressed->delta_array;
    opt_format.d_plain_deltas = compressed->d_plain_deltas;
    opt_format.num_partitions = compressed->num_partitions;
    opt_format.total_elements = compressed->total_values;

    // Call the optimized decompression kernel
    launchDecompressOptimized(opt_format, d_output, stream);
}

// Explicit instantiations for CompressedDataGLECO wrapper
template void launchDecompressOptimized<int32_t>(
    const CompressedDataGLECO<int32_t>*, int32_t*, hipStream_t);
template void launchDecompressOptimized<uint32_t>(
    const CompressedDataGLECO<uint32_t>*, uint32_t*, hipStream_t);
template void launchDecompressOptimized<int64_t>(
    const CompressedDataGLECO<int64_t>*, int64_t*, hipStream_t);
template void launchDecompressOptimized<uint64_t>(
    const CompressedDataGLECO<uint64_t>*, uint64_t*, hipStream_t);

// ============================================================================
// SIMPLE DECODER (for debugging/comparison with original)
// ============================================================================

/**
 * Simple grid-stride loop decoder, matching original L32.cu algorithm
 * This is NOT optimized, but should be bit-for-bit identical to encoder
 *
 * Uses CompressedDataOpt format passed by value (device-friendly)
 */
template<typename T>
__global__ void decompressPartitionsSimple(
    const CompressedDataOpt<T> compressed,
    T* __restrict__ output)
{
    __shared__ int32_t s_start;
    __shared__ int32_t s_len;
    __shared__ int32_t s_model_type;
    __shared__ int32_t s_delta_bits;
    __shared__ int64_t s_bit_offset_base;
    __shared__ double s_theta0, s_theta1;

    const int partition_idx = blockIdx.x;

    if (partition_idx >= compressed.num_partitions) return;

    // Load metadata
    if (threadIdx.x == 0) {
        s_start = compressed.d_start_indices[partition_idx];
        s_len = compressed.d_end_indices[partition_idx] - s_start;
        s_model_type = compressed.d_model_types[partition_idx];
        s_delta_bits = compressed.d_delta_bits[partition_idx];
        s_bit_offset_base = compressed.d_delta_array_bit_offsets[partition_idx];
        s_theta0 = compressed.d_model_params[partition_idx * 4];
        s_theta1 = compressed.d_model_params[partition_idx * 4 + 1];
    }
    __syncthreads();

    // Grid-stride loop (matching original)
    for (int local_idx = threadIdx.x; local_idx < s_len; local_idx += blockDim.x) {
        int global_idx = s_start + local_idx;
        if (global_idx >= compressed.total_elements) continue;

        T final_value;

        if (s_model_type == MODEL_DIRECT_COPY) {
            // Direct copy: extract full value
            if (s_delta_bits > 0) {
                int64_t bit_offset = s_bit_offset_base + (int64_t)local_idx * s_delta_bits;

                // Handle 64-bit values directly (for uint64_t) - must match encoder.cu:396-406
                if (s_delta_bits == 64 && sizeof(T) == 8) {
                    // Extract 64 bits from packed array using same logic as encoder
                    const int word_idx = bit_offset >> 5;
                    const int bit_in_word = bit_offset & 31;

                    uint64_t val64 = 0;
                    int bits_remaining = 64;
                    int current_word_idx = word_idx;
                    int current_bit_offset = bit_in_word;
                    int shift_amount = 0;

                    while (bits_remaining > 0) {
                        int bits_in_this_word = min(bits_remaining, 32 - current_bit_offset);
                        uint32_t word = compressed.delta_array[current_word_idx];
                        uint32_t mask = (bits_in_this_word == 32) ? ~0U : ((1U << bits_in_this_word) - 1U);
                        uint32_t extracted = (word >> current_bit_offset) & mask;

                        val64 |= (static_cast<uint64_t>(extracted) << shift_amount);

                        shift_amount += bits_in_this_word;
                        bits_remaining -= bits_in_this_word;
                        current_word_idx++;
                        current_bit_offset = 0;
                    }
                    final_value = static_cast<T>(val64);
                } else {
                    uint32_t extracted = extractDeltaDirect(compressed.delta_array, bit_offset, s_delta_bits);
                    final_value = static_cast<T>(extracted);
                }
            } else {
                final_value = static_cast<T>(0);
            }
        } else {
            // Model-based prediction
            double predicted = fma(s_theta1, static_cast<double>(local_idx), s_theta0);
            T pred_T = static_cast<T>(round(predicted));

            // Extract delta
            int32_t delta = 0;
            if (s_delta_bits > 0) {
                int64_t bit_offset = s_bit_offset_base + (int64_t)local_idx * s_delta_bits;
                delta = extractDeltaDirect(compressed.delta_array, bit_offset, s_delta_bits);
            }

            final_value = applyDelta(pred_T, static_cast<int64_t>(delta));
        }

        output[global_idx] = final_value;
    }
}

template<typename T>
void launchDecompressSimple(
    const CompressedDataGLECO<T>* compressed,
    T* d_output,
    hipStream_t stream)
{
    // Convert CompressedDataGLECO to CompressedDataOpt for device access
    CompressedDataOpt<T> opt_format;
    opt_format.d_start_indices = compressed->d_start_indices;
    opt_format.d_end_indices = compressed->d_end_indices;
    opt_format.d_model_types = compressed->d_model_types;
    opt_format.d_model_params = compressed->d_model_params;
    opt_format.d_delta_bits = compressed->d_delta_bits;
    opt_format.d_delta_array_bit_offsets = compressed->d_delta_array_bit_offsets;
    opt_format.delta_array = compressed->delta_array;
    opt_format.d_plain_deltas = compressed->d_plain_deltas;
    opt_format.num_partitions = compressed->num_partitions;
    opt_format.total_elements = compressed->total_values;

    const int num_blocks = compressed->num_partitions;
    const int threads_per_block = 256;

    decompressPartitionsSimple<T><<<num_blocks, threads_per_block, 0, stream>>>(
        opt_format, d_output);
}

// Explicit instantiations for simple decoder
template void launchDecompressSimple<int32_t>(const CompressedDataGLECO<int32_t>*, int32_t*, hipStream_t);
template void launchDecompressSimple<uint32_t>(const CompressedDataGLECO<uint32_t>*, uint32_t*, hipStream_t);
template void launchDecompressSimple<int64_t>(const CompressedDataGLECO<int64_t>*, int64_t*, hipStream_t);
template void launchDecompressSimple<uint64_t>(const CompressedDataGLECO<uint64_t>*, uint64_t*, hipStream_t);
