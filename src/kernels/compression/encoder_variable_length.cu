#include "hip/hip_runtime.h"
/**
 * Variable-Length Encoder for GLECO
 *
 * Implements adaptive partitioning based on data variance.
 * Extracted and adapted from L32.cu
 */

#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <vector>
#include <algorithm>
#include <iostream>
#include "L3_format.hpp"
#include "L3_codec.hpp"

// Constants
#define MIN_PARTITION_SIZE 128
#define PARTITION_MODEL_SIZE_BYTES 64.0

// ============================================================================
// Helper Functions
// ============================================================================

template<typename T>
__device__ __host__ inline bool mightOverflowDoublePrecision(T value) {
    if (std::is_signed<T>::value) {
        return false;
    } else {
        const double MAX_SAFE_DOUBLE = 9007199254740992.0; // 2^53
        return static_cast<double>(value) > MAX_SAFE_DOUBLE;
    }
}

template<typename T>
__device__ __host__ inline long long calculateDelta(T actual, T predicted) {
    if (std::is_signed<T>::value) {
        return static_cast<long long>(actual) - static_cast<long long>(predicted);
    } else {
        return static_cast<long long>(actual) - static_cast<long long>(predicted);
    }
}

__device__ double warpReduceSum(double val) {
    for (int offset = 16; offset > 0; offset >>= 1)
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

__device__ long long warpReduceMax(long long val) {
    for (int offset = 16; offset > 0; offset >>= 1)
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    return val;
}

__device__ double blockReduceSum(double val) {
    __shared__ double shared[32];
    int lane = threadIdx.x & 31;
    int wid = threadIdx.x >> 5;

    val = warpReduceSum(val);

    if (lane == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x < (blockDim.x >> 5)) ? shared[lane] : 0;
    if (wid == 0) val = warpReduceSum(val);

    return val;
}

__device__ long long blockReduceMax(long long val) {
    __shared__ long long shared[32];
    int lane = threadIdx.x & 31;
    int wid = threadIdx.x >> 5;

    val = warpReduceMax(val);

    if (lane == 0) shared[wid] = val;
    __syncthreads();

    val = (threadIdx.x < (blockDim.x >> 5)) ? shared[lane] : 0;
    if (wid == 0) val = warpReduceMax(val);

    return val;
}

// ============================================================================
// Variance Analysis Kernel
// ============================================================================

template<typename T>
__global__ void analyzeDataVarianceFast(
    const T* __restrict__ data,
    int data_size,
    int block_size,
    float* __restrict__ variances,
    int num_blocks) {

    for (int bid = blockIdx.x; bid < num_blocks; bid += gridDim.x) {
        int start = bid * block_size;
        int end = min(start + block_size, data_size);
        int n = end - start;

        if (n <= 0) continue;

        double sum = 0.0;
        double sum_sq = 0.0;
        double c1 = 0.0, c2 = 0.0;

        for (int i = start + threadIdx.x; i < end; i += blockDim.x) {
            double val = static_cast<double>(data[i]);

            // Kahan summation
            double y1 = val - c1;
            double t1 = sum + y1;
            c1 = (t1 - sum) - y1;
            sum = t1;

            double y2 = val * val - c2;
            double t2 = sum_sq + y2;
            c2 = (t2 - sum_sq) - y2;
            sum_sq = t2;
        }

        sum = warpReduceSum(sum);
        sum_sq = warpReduceSum(sum_sq);

        if ((threadIdx.x & 31) == 0) {
            atomicAdd(&variances[bid], static_cast<float>(sum_sq / n - (sum / n) * (sum / n)));
        }
    }
}

// ============================================================================
// Partition Creation Kernels
// ============================================================================

template<typename T>
__global__ void countPartitionsPerBlock(
    int data_size,
    int base_size,
    const float* __restrict__ variances,
    int num_variance_blocks,
    int* __restrict__ partition_counts_per_block,
    const float* __restrict__ variance_thresholds,
    const int* __restrict__ partition_sizes_for_buckets,
    int num_thresholds,
    int variance_block_multiplier)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= num_variance_blocks) return;

    float var = variances[i];
    int block_start = i * base_size * variance_block_multiplier;
    int block_end = min(block_start + base_size * variance_block_multiplier, data_size);

    int partition_size = partition_sizes_for_buckets[num_thresholds];
    for (int j = 0; j < num_thresholds; ++j) {
        if (var < variance_thresholds[j]) {
            partition_size = partition_sizes_for_buckets[j];
            break;
        }
    }

    int count = 0;
    if (partition_size > 0) {
        for (int j = block_start; j < block_end; j += partition_size) {
            if (j < data_size) {
                count++;
            }
        }
    }

    partition_counts_per_block[i] = count;
}

template<typename T>
__global__ void writePartitionsOrdered(
    int data_size,
    int base_size,
    const float* __restrict__ variances,
    int num_variance_blocks,
    const int* __restrict__ partition_offsets,
    int* __restrict__ partition_starts,
    int* __restrict__ partition_ends,
    const float* __restrict__ variance_thresholds,
    const int* __restrict__ partition_sizes_for_buckets,
    int num_thresholds,
    int variance_block_multiplier)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= num_variance_blocks) return;

    float var = variances[i];
    int block_start = i * base_size * variance_block_multiplier;
    int block_end = min(block_start + base_size * variance_block_multiplier, data_size);

    int partition_size = partition_sizes_for_buckets[num_thresholds];
    for (int j = 0; j < num_thresholds; ++j) {
        if (var < variance_thresholds[j]) {
            partition_size = partition_sizes_for_buckets[j];
            break;
        }
    }

    if (partition_size <= 0) return;

    int write_pos = partition_offsets[i];

    int local_idx = 0;
    for (int j = block_start; j < block_end; j += partition_size) {
        if (j < data_size) {
            partition_starts[write_pos + local_idx] = j;
            partition_ends[write_pos + local_idx] = min(j + partition_size, data_size);
            local_idx++;
        }
    }
}

// ============================================================================
// Model Fitting Kernel
// ============================================================================

template<typename T>
__global__ void fitPartitionsBatched_Optimized(
    const T* __restrict__ data,
    const int* __restrict__ partition_starts,
    const int* __restrict__ partition_ends,
    int* __restrict__ model_types,
    double* __restrict__ theta0_array,
    double* __restrict__ theta1_array,
    int* __restrict__ delta_bits_array,
    long long* __restrict__ max_errors,
    double* __restrict__ costs,
    int num_partitions)
{
    const int pid = blockIdx.x;
    if (pid >= num_partitions) {
        return;
    }

    __shared__ double s_theta0;
    __shared__ double s_theta1;
    __shared__ int s_has_overflow_flag;

    const int start = partition_starts[pid];
    const int end = partition_ends[pid];
    const int n = end - start;

    if (threadIdx.x == 0) {
        s_has_overflow_flag = false;
    }
    __syncthreads();

    if (n <= 0) {
        if (threadIdx.x == 0) {
            model_types[pid] = MODEL_DIRECT_COPY;
            costs[pid] = 0.0;
        }
        return;
    }

    bool local_overflow = false;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        if (mightOverflowDoublePrecision(data[start + i])) {
            local_overflow = true;
            break;
        }
    }

    if (local_overflow) {
        atomicExch(&s_has_overflow_flag, true);
    }
    __syncthreads();

    if (s_has_overflow_flag) {
        if (threadIdx.x == 0) {
            model_types[pid] = MODEL_DIRECT_COPY;
            theta0_array[pid] = 0.0;
            theta1_array[pid] = 0.0;
            delta_bits_array[pid] = sizeof(T) * 8;
            max_errors[pid] = 0;
            costs[pid] = PARTITION_MODEL_SIZE_BYTES + n * sizeof(T);
        }
        return;
    }

    // Fast linear regression
    double sum_x = 0.0, sum_y = 0.0, sum_xx = 0.0, sum_xy = 0.0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        double x = static_cast<double>(i);
        double y = static_cast<double>(data[start + i]);
        sum_x += x;
        sum_y += y;
        sum_xx = fma(x, x, sum_xx);
        sum_xy = fma(x, y, sum_xy);
    }

    sum_x = blockReduceSum(sum_x);
    sum_y = blockReduceSum(sum_y);
    sum_xx = blockReduceSum(sum_xx);
    sum_xy = blockReduceSum(sum_xy);

    if (threadIdx.x == 0) {
        double dn = static_cast<double>(n);
        double determinant = fma(dn, sum_xx, -(sum_x * sum_x));

        if (fabs(determinant) > 1e-10) {
            s_theta1 = fma(dn, sum_xy, -(sum_x * sum_y)) / determinant;
            s_theta0 = fma(-s_theta1, sum_x, sum_y) / dn;
        } else {
            s_theta1 = 0.0;
            s_theta0 = sum_y / dn;
        }
        model_types[pid] = MODEL_LINEAR;
        theta0_array[pid] = s_theta0;
        theta1_array[pid] = s_theta1;
    }
    __syncthreads();

    double theta0 = theta0_array[pid];
    double theta1 = theta1_array[pid];
    long long local_max_error = 0;

    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        double predicted = fma(theta1, static_cast<double>(i), theta0);
        T pred_T = static_cast<T>(round(predicted));
        long long delta = calculateDelta(data[start + i], pred_T);
        local_max_error = max(local_max_error, llabs(delta));
    }

    long long partition_max_error = blockReduceMax(local_max_error);

    if (threadIdx.x == 0) {
        max_errors[pid] = partition_max_error;

        int delta_bits = 0;
        if (partition_max_error > 0) {
            delta_bits = 64 - __clzll(static_cast<unsigned long long>(partition_max_error)) + 1;
        }
        delta_bits_array[pid] = delta_bits;

        double delta_array_bytes = static_cast<double>(n) * delta_bits / 8.0;
        costs[pid] = PARTITION_MODEL_SIZE_BYTES + delta_array_bytes;
    }
}

// ============================================================================
// Variable-Length Partitioner Class
// ============================================================================

template<typename T>
class GPUVariableLengthPartitionerV6 {
private:
    T* d_data;
    int data_size;
    int base_partition_size;
    hipStream_t stream;
    int variance_block_multiplier;
    int num_thresholds;

public:
    GPUVariableLengthPartitionerV6(const std::vector<T>& data,
                                   int base_size = 2048,
                                   hipStream_t hip_stream = 0,
                                   int multiplier = 8,
                                   int thresholds = 3)
        : data_size(data.size()),
          base_partition_size(base_size),
          stream(hip_stream),
          variance_block_multiplier(multiplier),
          num_thresholds(thresholds)
    {
        if (this->num_thresholds < 1) {
            this->num_thresholds = 1;
        }
        hipMalloc(&d_data, data_size * sizeof(T));
        hipMemcpyAsync(d_data, data.data(), data_size * sizeof(T),
                       hipMemcpyHostToDevice, stream);
    }

    ~GPUVariableLengthPartitionerV6() {
        if (d_data) hipFree(d_data);
    }

    std::vector<PartitionInfo> partition() {
        if (data_size == 0) return std::vector<PartitionInfo>();

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        int sm_count = prop.multiProcessorCount;

        int variance_block_size = base_partition_size * variance_block_multiplier;
        int num_variance_blocks = (data_size + variance_block_size - 1) / variance_block_size;
        float* d_variances;
        float* d_variance_thresholds;

        hipMalloc(&d_variances, num_variance_blocks * sizeof(float));
        hipMalloc(&d_variance_thresholds, num_thresholds * sizeof(float));
        hipMemsetAsync(d_variances, 0, num_variance_blocks * sizeof(float), stream);

        int threads = 128;
        int blocks = min(num_variance_blocks, sm_count * 4);

        analyzeDataVarianceFast<T><<<blocks, threads, 0, stream>>>(
            d_data, data_size, variance_block_size, d_variances, num_variance_blocks);

        thrust::device_ptr<float> var_ptr(d_variances);
        if (num_variance_blocks > 1) {
           thrust::sort(var_ptr, var_ptr + num_variance_blocks);
        }

        std::vector<float> h_thresholds(num_thresholds);
        for (int i = 0; i < num_thresholds; ++i) {
            long long idx = (long long)(i + 1) * num_variance_blocks / (num_thresholds + 1);
            if (idx >= num_variance_blocks) idx = num_variance_blocks - 1;
            if (idx < 0) idx = 0;
            h_thresholds[i] = (num_variance_blocks > 0) ? var_ptr[idx] : 0.0f;
        }

        hipMemcpyAsync(d_variance_thresholds, h_thresholds.data(),
                       num_thresholds * sizeof(float), hipMemcpyHostToDevice, stream);

        std::vector<int> h_partition_sizes_for_buckets(num_thresholds + 1);
        int min_partition_size_val = base_partition_size;
        for (int i = 0; i <= num_thresholds; ++i) {
            int shift = (num_thresholds / 2) - i;
            h_partition_sizes_for_buckets[i] = std::max(MIN_PARTITION_SIZE, base_partition_size << shift);
            if (h_partition_sizes_for_buckets[i] < min_partition_size_val) {
                min_partition_size_val = h_partition_sizes_for_buckets[i];
            }
        }

        int* d_partition_sizes_for_buckets;
        hipMalloc(&d_partition_sizes_for_buckets, (num_thresholds + 1) * sizeof(int));
        hipMemcpyAsync(d_partition_sizes_for_buckets, h_partition_sizes_for_buckets.data(),
                       (num_thresholds + 1) * sizeof(int), hipMemcpyHostToDevice, stream);

        int estimated_partitions = (min_partition_size_val > 0) ?
                                  (data_size / min_partition_size_val + 1) * 2 :
                                  data_size / MIN_PARTITION_SIZE;
        int* d_partition_starts;
        int* d_partition_ends;

        hipMalloc(&d_partition_starts, estimated_partitions * sizeof(int));
        hipMalloc(&d_partition_ends, estimated_partitions * sizeof(int));

        blocks = min((num_variance_blocks + threads - 1) / threads, sm_count * 2);

        int* d_partition_counts;
        int* d_partition_offsets;
        hipMalloc(&d_partition_counts, num_variance_blocks * sizeof(int));
        hipMalloc(&d_partition_offsets, (num_variance_blocks + 1) * sizeof(int));

        countPartitionsPerBlock<T><<<blocks, threads, 0, stream>>>(
            data_size, base_partition_size, d_variances, num_variance_blocks,
            d_partition_counts,
            d_variance_thresholds, d_partition_sizes_for_buckets,
            num_thresholds, variance_block_multiplier);

        thrust::device_ptr<int> counts_ptr(d_partition_counts);
        thrust::device_ptr<int> offsets_ptr(d_partition_offsets);
        thrust::exclusive_scan(counts_ptr, counts_ptr + num_variance_blocks, offsets_ptr);

        int h_num_partitions;
        int h_last_count;
        hipMemcpyAsync(&h_num_partitions,
                       d_partition_offsets + num_variance_blocks - 1,
                       sizeof(int), hipMemcpyDeviceToHost, stream);
        hipMemcpyAsync(&h_last_count,
                       d_partition_counts + num_variance_blocks - 1,
                       sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        h_num_partitions += h_last_count;

        if (h_num_partitions > estimated_partitions) {
            std::cerr << "Warning: Partition count " << h_num_partitions
                      << " exceeds estimate " << estimated_partitions << std::endl;
            h_num_partitions = estimated_partitions;
        }

        writePartitionsOrdered<T><<<blocks, threads, 0, stream>>>(
            data_size, base_partition_size, d_variances, num_variance_blocks,
            d_partition_offsets,
            d_partition_starts, d_partition_ends,
            d_variance_thresholds, d_partition_sizes_for_buckets,
            num_thresholds, variance_block_multiplier);

        hipFree(d_partition_counts);
        hipFree(d_partition_offsets);

        int* d_model_types;
        double* d_theta0;
        double* d_theta1;
        int* d_delta_bits;
        long long* d_max_errors;
        double* d_costs;

        hipMalloc(&d_model_types, h_num_partitions * sizeof(int));
        hipMalloc(&d_theta0, h_num_partitions * sizeof(double));
        hipMalloc(&d_theta1, h_num_partitions * sizeof(double));
        hipMalloc(&d_delta_bits, h_num_partitions * sizeof(int));
        hipMalloc(&d_max_errors, h_num_partitions * sizeof(long long));
        hipMalloc(&d_costs, h_num_partitions * sizeof(double));

        int threads_per_block = 256;
        int grid_size = h_num_partitions;

        if (grid_size > 0) {
            fitPartitionsBatched_Optimized<T><<<grid_size, threads_per_block, 0, stream>>>(
                d_data,
                d_partition_starts,
                d_partition_ends,
                d_model_types,
                d_theta0,
                d_theta1,
                d_delta_bits,
                d_max_errors,
                d_costs,
                h_num_partitions
            );
        }

        std::vector<int> h_starts(h_num_partitions);
        std::vector<int> h_ends(h_num_partitions);
        std::vector<int> h_model_types(h_num_partitions);
        std::vector<double> h_theta0(h_num_partitions);
        std::vector<double> h_theta1(h_num_partitions);
        std::vector<int> h_delta_bits(h_num_partitions);
        std::vector<long long> h_max_errors(h_num_partitions);

        if (h_num_partitions > 0) {
            hipMemcpyAsync(h_starts.data(), d_partition_starts,
                           h_num_partitions * sizeof(int), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_ends.data(), d_partition_ends,
                           h_num_partitions * sizeof(int), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_model_types.data(), d_model_types,
                           h_num_partitions * sizeof(int), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_theta0.data(), d_theta0,
                           h_num_partitions * sizeof(double), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_theta1.data(), d_theta1,
                           h_num_partitions * sizeof(double), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_delta_bits.data(), d_delta_bits,
                           h_num_partitions * sizeof(int), hipMemcpyDeviceToHost, stream);
            hipMemcpyAsync(h_max_errors.data(), d_max_errors,
                           h_num_partitions * sizeof(long long), hipMemcpyDeviceToHost, stream);
        }
        hipStreamSynchronize(stream);

        std::vector<PartitionInfo> result;
        result.reserve(h_num_partitions);

        for (int i = 0; i < h_num_partitions; i++) {
            PartitionInfo info;
            info.start_idx = h_starts[i];
            info.end_idx = h_ends[i];
            info.model_type = h_model_types[i];
            info.model_params[0] = h_theta0[i];
            info.model_params[1] = h_theta1[i];
            info.model_params[2] = 0.0;
            info.model_params[3] = 0.0;
            info.delta_bits = h_delta_bits[i];
            info.delta_array_bit_offset = 0;
            info.error_bound = h_max_errors[i];
            result.push_back(info);
        }

        if (!result.empty()) {
            std::sort(result.begin(), result.end(),
                     [](const PartitionInfo& a, const PartitionInfo& b) {
                         return a.start_idx < b.start_idx;
                     });

            result[0].start_idx = 0;
            result.back().end_idx = data_size;

            for (size_t i = 0; i < result.size() - 1; i++) {
                if (result[i].end_idx != result[i + 1].start_idx) {
                    result[i].end_idx = result[i + 1].start_idx;
                }
            }
        }

        // Cleanup
        hipFree(d_variances);
        hipFree(d_variance_thresholds);
        hipFree(d_partition_sizes_for_buckets);
        hipFree(d_partition_starts);
        hipFree(d_partition_ends);
        hipFree(d_model_types);
        hipFree(d_theta0);
        hipFree(d_theta1);
        hipFree(d_delta_bits);
        hipFree(d_max_errors);
        hipFree(d_costs);

        return result;
    }
};

// ============================================================================
// Public API
// ============================================================================

template<typename T>
std::vector<PartitionInfo> createPartitionsVariableLength(
    const std::vector<T>& data,
    int base_partition_size,
    int* num_partitions_out,
    hipStream_t stream,
    int variance_block_multiplier,
    int num_thresholds)
{
    GPUVariableLengthPartitionerV6<T> partitioner(
        data, base_partition_size, stream,
        variance_block_multiplier, num_thresholds);

    std::vector<PartitionInfo> result = partitioner.partition();

    if (num_partitions_out) {
        *num_partitions_out = static_cast<int>(result.size());
    }

    return result;
}

// Explicit template instantiation
template class GPUVariableLengthPartitionerV6<uint64_t>;
template class GPUVariableLengthPartitionerV6<int64_t>;
template class GPUVariableLengthPartitionerV6<uint32_t>;
template class GPUVariableLengthPartitionerV6<int32_t>;

template std::vector<PartitionInfo> createPartitionsVariableLength<uint64_t>(
    const std::vector<uint64_t>&, int, int*, hipStream_t, int, int);
template std::vector<PartitionInfo> createPartitionsVariableLength<int64_t>(
    const std::vector<int64_t>&, int, int*, hipStream_t, int, int);
template std::vector<PartitionInfo> createPartitionsVariableLength<uint32_t>(
    const std::vector<uint32_t>&, int, int*, hipStream_t, int, int);
template std::vector<PartitionInfo> createPartitionsVariableLength<int32_t>(
    const std::vector<int32_t>&, int, int*, hipStream_t, int, int);
