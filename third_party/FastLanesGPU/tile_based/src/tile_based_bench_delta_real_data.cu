#include "hip/hip_runtime.h"
#include "config.hpp"
#include "cub/util_debug.cuh"
#include "kernel.cuh"
#include "binpack_kernel.cuh"
#include "utils/gpu_utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <fstream>
#include <cstring>

uint deltaBinPack(int*& in, int*& out, uint*& block_offsets, uint num_entries) {
	uint offset = 0;

	uint block_size      = 128;
	uint elem_per_thread = 4;
	uint tile_size       = block_size * elem_per_thread;

	uint miniblock_count = 4;
	uint total_count     = num_entries;
	uint first_val       = in[0];

	out[0] = block_size;
	out[1] = miniblock_count;
	out[2] = total_count;
	out[3] = first_val;

	offset += 4;

	for (uint tile_start = 0; tile_start < num_entries; tile_start += tile_size) {
		uint block_index   = tile_start / block_size;
		int  tmp_first_val = in[0];

		out[offset] = tmp_first_val;
		offset++;

		// Compute the deltas
		for (int i = tile_size - 1; i > 0; i--) {
			in[i] = in[i] - in[i - 1];
		}
		in[0] = 0;

		for (int block_start = 0; block_start < block_size * 4; block_start += block_size, block_index += 1) {
			block_offsets[block_index] = offset;

			// For FOR - Find min val
			int min_val = in[0];
			for (int i = 1; i < block_size; i++) {
				if (in[i] < min_val) { min_val = in[i]; }
			}

			min_val = 0; /* HACK */
			for (int i = 0; i < block_size; i++) {
				in[i] = in[i] - min_val;
			}

			out[offset] = min_val;
			offset++;

			// Subtracting min_val ensures that all input vals are >= 0
			// Going forward in and out will both be treated as unsigned integers.
			uint* inp  = (uint*)in;
			uint* outp = (uint*)out;

			uint  miniblock_size      = block_size / miniblock_count;
			uint* miniblock_bitwidths = new uint[miniblock_count];
			for (int i = 0; i < miniblock_count; i++) {
				miniblock_bitwidths[i] = 0;
			}

			for (uint miniblock = 0; miniblock < miniblock_count; miniblock++) {
				for (uint i = 0; i < miniblock_size; i++) {
					uint bitwidth = uint(ceil(log2(inp[miniblock * miniblock_size + i] + 1)));
					if (bitwidth > miniblock_bitwidths[miniblock]) { miniblock_bitwidths[miniblock] = bitwidth; }
				}
			}

			// Extra for Simple BinPack
			uint max_bitwidth = miniblock_bitwidths[0];
			for (int i = 1; i < miniblock_count; i++) {
				max_bitwidth = max(max_bitwidth, miniblock_bitwidths[i]);
			}
			for (int i = 0; i < miniblock_count; i++) {
				miniblock_bitwidths[i] = max_bitwidth;
			}
			outp[offset] = miniblock_bitwidths[0] + (miniblock_bitwidths[1] << 8) + (miniblock_bitwidths[2] << 16) +
			               (miniblock_bitwidths[3] << 24);
			offset++;

			for (int miniblock = 0; miniblock < miniblock_count; miniblock++) {
				uint bitwidth = miniblock_bitwidths[miniblock];
				uint shift    = 0;
				for (int i = 0; i < miniblock_size; i++) {
					if (shift + bitwidth > 32) {
						if (shift != 32) { outp[offset] += inp[miniblock * miniblock_size + i] << shift; }
						offset++;
						shift        = (shift + bitwidth) & (32 - 1);
						outp[offset] = inp[miniblock * miniblock_size + i] >> (bitwidth - shift);
					} else {
						outp[offset] += inp[miniblock * miniblock_size + i] << shift;
						shift += bitwidth;
					}
				}
				offset++;
			}

			// Increment the input pointer by block size
			in += block_size;
		}
	}

	block_offsets[num_entries / block_size] = offset;

	return offset;
}

template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void runDBinKernel(int* col, uint* col_block_start, uint* col_data, int num_entries) {
	int tile_size   = BLOCK_THREADS * ITEMS_PER_THREAD;
	int tile_idx    = blockIdx.x;
	int tile_offset = tile_idx * tile_size;

	// Load a segment of consecutive items that are blocked across threads
	int col_block[ITEMS_PER_THREAD];

	int  num_tiles      = (num_entries + tile_size - 1) / tile_size;
	int  num_tile_items = tile_size;
	bool is_last_tile   = false;
	if (tile_idx == num_tiles - 1) {
		num_tile_items = num_entries - tile_offset;
		is_last_tile   = true;
	}

	extern __shared__ uint shared_buffer[];
	LoadDBinPack<BLOCK_THREADS, ITEMS_PER_THREAD>(
	    col_block_start, col_data, shared_buffer, col_block, is_last_tile, num_tile_items);

	__syncthreads();

	for (int i = 0; i < ITEMS_PER_THREAD; i++) {
		col[tile_size * tile_idx + i * 128 + threadIdx.x] = col_block[i];
	}
}

namespace tile_based {
template <typename T>
T* loadColumnToGPU(T* src, int len) {
	T* dest = nullptr;
	hipMalloc((void**)&dest, sizeof(T) * len);
	CubDebugExit(hipMemcpy(dest, src, sizeof(T) * len, hipMemcpyHostToDevice));
	return dest;
}

}

uint64_t load_data_from_file(const char* filename, uint32_t** data, uint64_t max_elements) {
	std::string fname(filename);
	uint64_t count = 0;

	if (fname.find(".bin") != std::string::npos) {
		// Binary file
		FILE* f = fopen(filename, "rb");
		if (!f) {
			std::cerr << "Cannot open file: " << filename << std::endl;
			return 0;
		}

		fseek(f, 0, SEEK_END);
		long file_size = ftell(f);
		fseek(f, 0, SEEK_SET);

		// Check if uint32 or uint64
		if (fname.find("uint32") != std::string::npos) {
			count = std::min((uint64_t)(file_size / sizeof(uint32_t)), max_elements);
			*data = new uint32_t[count];
			fread(*data, sizeof(uint32_t), count, f);
		} else {
			// uint64 - convert to uint32
			count = std::min((uint64_t)(file_size / sizeof(uint64_t)), max_elements);
			uint64_t* temp = new uint64_t[count];
			fread(temp, sizeof(uint64_t), count, f);
			*data = new uint32_t[count];
			for (uint64_t i = 0; i < count; i++) {
				(*data)[i] = (uint32_t)temp[i];
			}
			delete[] temp;
		}
		fclose(f);
	} else {
		// Text file
		std::ifstream file(filename);
		if (!file.is_open()) {
			std::cerr << "Cannot open file: " << filename << std::endl;
			return 0;
		}

		std::vector<uint32_t> temp_data;
		uint64_t value;
		while (file >> value && temp_data.size() < max_elements) {
			temp_data.push_back((uint32_t)value);
		}
		file.close();

		count = temp_data.size();
		*data = new uint32_t[count];
		memcpy(*data, temp_data.data(), count * sizeof(uint32_t));
	}

	return count;
}

int main(int argc, char** argv) {

	if (argc < 2) {
		std::cout << "Usage: " << argv[0] << " <data_file> [dataset_name]" << std::endl;
		return 1;
	}

	hipSetDevice(0);

	const char* data_file = argv[1];
	const char* dataset_name = (argc > 2) ? argv[2] : "unknown";

	std::cout << "Dataset: " << dataset_name << std::endl;
	std::cout << "Loading data from: " << data_file << std::endl;

	uint32_t* h_org_arr = nullptr;
	uint64_t n_tup = load_data_from_file(data_file, &h_org_arr, 1 << 28);  // Max 256M elements

	if (n_tup == 0) {
		std::cerr << "Failed to load data" << std::endl;
		return 1;
	}

	// Round down to multiple of 512 (tile_size)
	n_tup = (n_tup / 512) * 512;

	std::cout << "Loaded " << n_tup << " elements (" << (n_tup * sizeof(uint32_t)) / (1024.0 * 1024.0) << " MB)" << std::endl;

	int       block_size       = 128;
	int       elem_per_thread  = 4;
	int       tile_size        = block_size * elem_per_thread;
	int       num_blocks       = n_tup / block_size;
	auto*     encoded_data     = new int[n_tup * 2]();  // Extra space
	uint64_t  ofs_c            = num_blocks + 1;
	auto*     ofs_arr          = new uint[ofs_c]();
	auto*     copy_data        = new int[n_tup];
	const int num_threads      = 128;
	const int items_per_thread = 4;
	int*      col              = nullptr;
	size_t    dg               = (n_tup + tile_size - 1) / tile_size;
	size_t    db               = num_threads;
	size_t    ns               = 3000;
	int*      temp             = new int[n_tup];

	hipMalloc((void**)&col, n_tup * sizeof(int));

	// Copy to int array
	memcpy(copy_data, h_org_arr, n_tup * sizeof(int));

	// Measure encoding time
	auto encode_start = std::chrono::high_resolution_clock::now();
	uint encoded_size = deltaBinPack(copy_data, encoded_data, ofs_arr, n_tup);
	auto encode_end = std::chrono::high_resolution_clock::now();
	double encode_time_ms = std::chrono::duration<double, std::milli>(encode_end - encode_start).count();

	tile_based::encoded_column h_col {ofs_arr, reinterpret_cast<uint*>(encoded_data), n_tup * 4};

	// Measure H2D transfer time
	hipEvent_t h2d_start, h2d_stop;
	hipEventCreate(&h2d_start);
	hipEventCreate(&h2d_stop);

	hipEventRecord(h2d_start);
	uint* d_col_block_start = tile_based::loadColumnToGPU<uint>(h_col.block_start, num_blocks + 1);
	uint* d_col_data        = tile_based::loadColumnToGPU<uint>(h_col.data, encoded_size);
	hipEventRecord(h2d_stop);
	hipEventSynchronize(h2d_stop);

	float h2d_time_ms;
	hipEventElapsedTime(&h2d_time_ms, h2d_start, h2d_stop);

	tile_based::encoded_column d_col {d_col_block_start, d_col_data};

	hipDeviceSynchronize();

	runDBinKernel<num_threads, items_per_thread><<<dg, db, ns>>>(col, d_col.block_start, d_col.data, n_tup);

	// Measure D2H transfer time
	hipEvent_t d2h_start, d2h_stop;
	hipEventCreate(&d2h_start);
	hipEventCreate(&d2h_stop);

	hipEventRecord(d2h_start);
	CubDebugExit(hipMemcpy(temp, col, sizeof(int) * n_tup, hipMemcpyDeviceToHost));
	hipEventRecord(d2h_stop);
	hipEventSynchronize(d2h_stop);

	float d2h_time_ms;
	hipEventElapsedTime(&d2h_time_ms, d2h_start, d2h_stop);

	// Verify correctness
	bool correct = true;
	for (uint64_t i = 0; i < n_tup; i++) {
		if (h_org_arr[i] != (uint32_t)temp[i]) {
			std::cout << "ERROR at " << i << ": " << h_org_arr[i] << " != " << temp[i] << '\n';
			correct = false;
			break;
		}
	}
	if (correct) {
		std::cout << "✓ Verification passed!" << '\n';
	}

	// Run decode trials
	int num_trials = 10;
	float query_time;
	SETUP_TIMING();

	// Warmup
	for (int t = 0; t < 3; t++) {
		runDBinKernel<num_threads, items_per_thread><<<dg, db, ns>>>(col, d_col.block_start, d_col.data, n_tup);
		hipDeviceSynchronize();
	}

	float total_decode_time = 0.0f;
	for (int t = 0; t < num_trials; t++) {
		hipEventRecord(start, nullptr);
		runDBinKernel<num_threads, items_per_thread><<<dg, db, ns>>>(col, d_col.block_start, d_col.data, n_tup);
		hipEventRecord(stop, nullptr);

		hipEventSynchronize(stop);
		hipEventElapsedTime(&query_time, start, stop);
		total_decode_time += query_time;

		CubDebugExit(hipPeekAtLastError());
		CubDebugExit(hipDeviceSynchronize());
	}

	float avg_decode_time = total_decode_time / num_trials;

	// Calculate metrics
	size_t original_size_bytes = n_tup * sizeof(uint32_t);
	size_t compressed_size_bytes = encoded_size * sizeof(uint32_t);
	double compression_ratio = (double)original_size_bytes / (double)compressed_size_bytes;
	double original_size_mb = original_size_bytes / (1024.0 * 1024.0);
	double compressed_size_mb = compressed_size_bytes / (1024.0 * 1024.0);
	double decode_throughput_gbps = (original_size_bytes / (1024.0 * 1024.0 * 1024.0)) / (avg_decode_time / 1000.0);

	// Output JSON format for easy parsing
	std::cout << "\n========== BENCHMARK RESULTS JSON ==========" << std::endl;
	std::cout << "{" << std::endl;
	std::cout << "  \"dataset\": \"" << dataset_name << "\"," << std::endl;
	std::cout << "  \"algorithm\": \"Delta\"," << std::endl;
	std::cout << "  \"data_size_mb\": " << original_size_mb << "," << std::endl;
	std::cout << "  \"compressed_size_mb\": " << compressed_size_mb << "," << std::endl;
	std::cout << "  \"compression_ratio\": " << compression_ratio << "," << std::endl;
	std::cout << "  \"encode_time_ms\": " << encode_time_ms << "," << std::endl;
	std::cout << "  \"h2d_transfer_ms\": " << h2d_time_ms << "," << std::endl;
	std::cout << "  \"decode_time_ms\": " << avg_decode_time << "," << std::endl;
	std::cout << "  \"d2h_transfer_ms\": " << d2h_time_ms << "," << std::endl;
	std::cout << "  \"decode_throughput_gbps\": " << decode_throughput_gbps << std::endl;
	std::cout << "}" << std::endl;
	std::cout << "===========================================" << std::endl;

	// Cleanup
	delete[] h_org_arr;
	delete[] encoded_data;
	delete[] ofs_arr;
	delete[] copy_data;
	delete[] temp;
	hipFree(col);
	hipFree(d_col_block_start);
	hipFree(d_col_data);
}
